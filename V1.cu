#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "auxiliary.h"
#include <math.h>
#include <stdbool.h>

#define PIXELS 64 // meaning image with PIXELSxPIXELS pixels
#define filtSigma 0.02
#define patchSize 3
#define patchSigma 3
__device__ void comparison(float *comparison,int i, int j, float *G, float *image);
__device__ void compute_weight(float *w, int i, int j, float *G, float *image);
__host__ float * gaussian();
__global__ void compute_f_pixel(float *f_image,float *image, int padded_size, float *G);

//! global variables
__device__ const int dev_PIXELS = PIXELS;
__device__ const float dev_filtSigma = (float)filtSigma;
__device__ const int dev_patchSize = patchSize;
__device__ const int dev_padding = patchSize/2;



__host__ float *nonLocalMeans(float *host_image){

/*
     ##############################
                V1 START
     ##############################
*/

	int padding = patchSize/2;
    int n = PIXELS * PIXELS;
	int padded_size = n + 4*(padding*PIXELS+ padding *padding);
	
	// Creation of Gaussian Filter
    float *G;
	hipMallocManaged(&G, patchSize*patchSize*sizeof(float));
	if(G == NULL){
        printf("Error.\n");
        exit(1);
    }
	float *temp_G = gaussian();
	memcpy(G,temp_G,patchSize*patchSize*sizeof(float));
	
	// Have a copy of the original in cuda memory
	float *image;
	hipMallocManaged(&image, padded_size*sizeof(float));
	if(image == NULL){
        printf("Error.\n");
        exit(1);
    }
	memcpy(image,host_image, padded_size*sizeof(float));

	// Creation of filtered image C array
	float *f_image;
	hipMallocManaged(&f_image, padded_size*sizeof(float));
	if(f_image == NULL){
        printf("Error.\n");
        exit(1);
    }
	for(int i=0; i<padded_size; i++)
		f_image[i]=(float)-1;
	
	
	// Creation of the kernel
    compute_f_pixel<<<PIXELS,PIXELS>>>(f_image,image, padded_size, G);
	hipDeviceSynchronize();
	
	hipFree(G);
	free(temp_G);
	hipFree(image);
/*
     ##############################
                 V1 END
     ##############################
*/
    return f_image;
}


__global__ void compute_f_pixel(float *f_image,float *image, int padded_size, float *G){
	
	// index i is calculated so that it iterates the original image minus the padding
	int i = blockIdx.x*(blockDim.x+2*dev_padding)+(threadIdx.x+dev_padding) + dev_padding*dev_PIXELS+2*dev_padding*dev_padding;
	
	if(i < padded_size){
		f_image[i] = 0;
		float Z = 0;
		float w;
		
		for(int r=dev_padding; r<dev_PIXELS+dev_padding; r++){
            for(int j=dev_padding; j<dev_PIXELS+dev_padding; j++){
				compute_weight(&w, i, r*(dev_PIXELS+2*dev_padding)+j, G, image);
				Z += w;
				f_image[i] += w * image[r*(dev_PIXELS+2*dev_padding)+j];
			}
		}
		f_image[i] = f_image[i] / Z; 
	}
}

__host__ int main(){
	
	// Convert txt to C array
	int padding = patchSize/2;
    float *host_image = read_txt(PIXELS, padding);
	float *f_image;
	hipMallocManaged(&f_image,0);
	
	
    struct timespec tic;
    clock_gettime( CLOCK_MONOTONIC, &tic);

	// Non-Local-Means
    f_image = nonLocalMeans(host_image);
	
    struct timespec toc;
    clock_gettime( CLOCK_MONOTONIC, &toc);
    printf("\n   ******************************\n     V1 duration = %f sec\n   ******************************\n\n",time_spent(tic, toc));

	
	// Convert C array to txt
	int padded_size = PIXELS*PIXELS + 4*(padding*PIXELS+ padding *padding);
    FILE *f = fopen("filtered_image_V1.txt", "w");
    int counter = 0;
    for(int i=(PIXELS*padding+2*padding*padding); i<padded_size - (PIXELS*padding+2*padding*padding); i++){
        if(f_image[i] == (float)-1)
            continue;
        fprintf(f, "%f ", f_image[i]);
        counter++;
        if(counter == PIXELS){
            counter = 0;
            fprintf(f, "\n");
        }
    }
    fclose(f);
    free(host_image);
	hipFree(f_image);
    return 0;
}


//! Compares Patch i and Patch j
__device__ void comparison(float *comparison_value,int i, int j, float *G, float *image){
    for(int k=0; k<dev_patchSize; k++){
        for(int l=0; l<dev_patchSize; l++){
            if(image[i+(k-dev_padding)*(dev_PIXELS+2*dev_padding)+  l-dev_padding] != (float)-1 && image[j+(k-dev_padding)*(dev_PIXELS+2*dev_padding) + l-dev_padding] != (float)-1){
                float diff = image[i+(k-dev_padding)*(dev_PIXELS+2*dev_padding)+  l-dev_padding] - image[j+(k-dev_padding)*(dev_PIXELS+2*dev_padding) + l-dev_padding];
                *comparison_value += G[k*dev_patchSize+l] * diff * diff;
            }
        }
    }
}


//! Computes the w(i,j)
__device__ void compute_weight(float *w, int i, int j, float *G, float *image){
	float comparison_value = 0;
	comparison(&comparison_value, i, j, G, image);
	*w = (float)(exp(-comparison_value/(dev_filtSigma*dev_filtSigma)));
}

//! Compute the gaussian filter
__host__ float * gaussian(){
    float *G = (float *)malloc(patchSize*patchSize*sizeof(float));
	if(G == NULL){
            printf("Error.\n");
            exit(1);
        }
    int bound = patchSize/2;
    for(int x=-bound; x<=bound; x++){ // if patchSize=5 then x = -2,-1,0,+1,+2
        for(int y=-bound; y<=bound; y++){
            float G_temp = exp(-(float)(x*x+y*y)/(float)(2*patchSigma*patchSigma))/(float)(2*M_PI*patchSigma*patchSigma); // 2D Gaussian filter
            int i = (x+bound)*patchSize +(y+bound); // i = 0*5+{0,1,2,3,4}, 1*5+{0,1,2,3,4},..., 4*5+{0,1,2,3,4}
            G[i] = G_temp;
        }
    }
    return G;
}

